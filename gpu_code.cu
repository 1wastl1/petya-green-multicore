#include "hip/hip_runtime.h"
 #include <hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include <map>

#include "petya.h"

#define SHL(x, s) ((uint32_t) ((x) << ((s) & 31)))
#define SHR(x, s) ((uint32_t) ((x) >> (32 - ((s) & 31))))
#define ROTL(x, s) ((uint32_t) (SHL((x), (s)) | SHR((x), (s))))

#define NR_THREADS 1024
#define NR_BLOCKS 1

#define NR_KEYS (long)(NR_THREADS*NR_BLOCKS)

using namespace std;


// -- 16 Byte Key funtions...

void calculate16ByteKeyFromIndex(unsigned long index, char*key) {
	// cc??cc??cc??cc??
	
	char keyChars[] = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
	// char key[17];
	
	memset(key, 'x', 16);
	  
	int posToKey[] = {13,12,9,8,5,4,1,0};
	  
	for (int i=0; i<8; i++) {
		int characterNumber = index % (26*2+10);
		key[posToKey[i]] = keyChars[characterNumber];					
		index /= (26*2+10);
	}
		
	// key[16] = 0;
	
	//char * result = (char *)malloc(17);
	//printf("Key calculated from index is: %s\r\n", &key[0]);
	// memcpy(result, key, 17);
	//return result;
} 

void nextKey16Byte(char *key) {
	char keyChars[] = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
	map<char, int> keyToIndexMap;
	
	
	for (int i=0; i<sizeof(keyChars); i++) {
		keyToIndexMap[keyChars[i]]=i;
	}		
	
	int posToKey[] = {13,12,9,8,5,4,1,0};
	
	for (int i=0; i<8; i++) {
		int idx = keyToIndexMap[key[posToKey[i]]];		
		idx++;
		idx %=sizeof(keyChars);
		key[posToKey[i]] = keyChars[idx];
		
		if (idx!=0) break;
	}
	
	printf("Next key of is %s\r\n",key);
}


__global__ void gpu_crypt_and_validate(uint8_t *keys,
                           
                            uint8_t nonce[8],
                            uint32_t si,
                            uint8_t *buf,
                            uint32_t buflen,
                            bool *isValid,
							int nrTotal,
							unsigned long nrKeysToCalculatePerThread)
{
  // This is initialized here to save time to calculate next key later, todo: Calculate outside and provide with parameters...
  char keyChars[] = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
    
  int keyToIndexMap[256];
  for (int i=0; i<sizeof(keyChars);i++) {
	  for (int j=0; j<256;j++) {
		  if (keyChars[i]==(char)j) {
			  keyToIndexMap[(char)j]=i;
		  }
	  }
  }

  int threadNr = blockDim.x * blockIdx.x + threadIdx.x;

  if (threadNr>=nrTotal) return;

  uint8_t *key = keys + (threadNr*(KEY_SIZE));   
  
  while (nrKeysToCalculatePerThread>0) {
	  (isValid)[threadNr] = false;

	  uint8_t keystream[64];
	  uint8_t n[16] = { 0 };
	  uint32_t i;

	  for (i = 0; i < 8; ++i)
		n[i] = nonce[i];

	  if (si % 64 != 0) {
		// s20_rev_littleendian(n+8, si / 64);
		(n+8)[0] = (si / 64);
		(n+8)[1] = (si / 64)>>8;
		(n+8)[2] = (si / 64)>>16;
		(n+8)[3] = (si / 64)>>24;

		// --------------------------------
		// s20_expand16(key, n, keystream);
		// --------------------------------

		  int i, j;
	  uint8_t t[4][4] = {
		{ 'e', 'x', 'p', 'a' },
		{ 'n', 'd', ' ', '1' },
		{ '6', '-', 'b', 'y' },
		{ 't', 'e', ' ', 'k' }
	  };

	  for (i = 0; i < 64; i += 20)
		for (j = 0; j < 4; ++j)
		  keystream[i + j] = t[i / 20][j];
	
	  for (i = 0; i < 16; ++i) {
		keystream[4+i]  = key[i];
		keystream[44+i] = key[i];
		keystream[24+i] = n[i];
	  }

	  // ____________________
	  // s20_hash(keystream);
	  // --------------------

	//    int i;
	  uint32_t x[16];
	  uint32_t z[16];
	
	  for (i = 0; i < 16; ++i) {

		// s20_littleendian
		uint8_t* result = keystream + (4 * i);
		x[i] = z[i] = (int16_t)(result[0]+(result[1]<<8)); //  s20_littleendian(seq + (4 * i));
	  }
	
	  for (i = 0; i < 10; ++i) {
	//    s20_doubleround(z);

	  // ColumnRound
	  // s20_quarterround(&x[0], &x[4], &x[8], &x[12]);

	  z[4] =  z[4]  ^ ROTL(z[0]  + z[12], 7);
	  z[8] =  z[8]  ^ ROTL(z[4]  + z[0], 9);
	  z[12] = z[12] ^ ROTL(z[8]  + z[4], 13);
	  z[0] =  z[0]  ^ ROTL(z[12] + z[8], 18);

	  // s20_quarterround(&x[5], &x[9], &x[13], &x[1]);
	  z[9] =  z[9]  ^ ROTL(z[5]  + z[1], 7);
	  z[13] = z[13] ^ ROTL(z[9]  + z[5], 9);
	  z[1] =  z[1]  ^ ROTL(z[13] + z[9], 13);
	  z[5] =  z[5]  ^ ROTL(z[1]  + z[13], 18);

	  // s20_quarterround(&x[10], &x[14], &x[2], &x[6]);
	  z[14]=  z[14] ^ ROTL(z[10] + z[6], 7);
	  z[2] =  z[2]  ^ ROTL(z[14] + z[10], 9);
	  z[6] =  z[6]  ^ ROTL(z[2]  + z[14], 13);
	  z[10] = z[10] ^ ROTL(z[6]  + z[2], 18);

	  // s20_quarterround(&x[15], &x[3], &x[7], &x[11]);
	  z[3] =  z[3]  ^ ROTL(z[15] + z[11], 7);
	  z[7] =  z[7]  ^ ROTL(z[3]  + z[15], 9);
	  z[11] = z[11] ^ ROTL(z[7]  + z[3], 13);
	  z[15] = z[15] ^ ROTL(z[11] + z[7], 18);

	  // Rowround
	  // s20_quarterround(&y[0], &y[1], &y[2], &y[3]);
	  z[1] = z[1] ^ ROTL(z[0]+  z[3], 7);
	  z[2] = z[2] ^ ROTL(z[1] + z[0], 9);
	  z[3] = z[3] ^ ROTL(z[2] + z[1], 13);
	  z[0] = z[0] ^ ROTL(z[3] + z[2], 18);

	  // s20_quarterround(&y[5], &y[6], &y[7], &y[4]);
	  z[6] = z[6] ^ ROTL(z[5] + z[4], 7);
	  z[7] = z[7] ^ ROTL(z[6] + z[5], 9);
	  z[4] = z[4] ^ ROTL(z[7] + z[6], 13);
	  z[5] = z[5] ^ ROTL(z[4] + z[7], 18);

	  // s20_quarterround(&y[10], &y[11], &y[8], &y[9]);
	  z[11] = z[11] ^ ROTL(z[10] + z[9], 7);
	  z[8] =  z[8]  ^ ROTL(z[11] + z[10], 9);
	  z[9] =  z[9]  ^ ROTL(z[8] +  z[11], 13);
	  z[10] = z[10] ^ ROTL(z[9] +  z[8], 18);
	  
	  // s20_quarterround(&y[15], &y[12], &y[12], &y[14]);
	  z[12] = z[12] ^ ROTL(z[15] + z[14], 7);
	  z[12] = z[12] ^ ROTL(z[12] + z[15], 9);
	  z[14] = z[14] ^ ROTL(z[12] + z[12], 13);
	  z[15] = z[15] ^ ROTL(z[14] + z[12], 18);
	  }
	
	  for (i = 0; i < 16; ++i) {
		z[i] += x[i];
		// s20_rev_littleendian(seq + (4 * i), z[i]);
		  (keystream + (4 * i))[0] = z[i];
		  (keystream + (4 * i))[1] = z[i] >> 8;
		  (keystream + (4 * i))[2] = z[i] >> 16;
		  (keystream + (4 * i))[3] = z[i] >> 24;
	  }
	  


	  }

	  for (int bufPos = 0; bufPos < buflen; ++bufPos) {
		if ((si + bufPos) % 64 == 0) {
		  //s20_rev_littleendian(n+8, ((si + i) / 64));
		  (n+8)[0] = ((si + bufPos) / 64);
		  (n+8)[1] = ((si + bufPos) / 64)>>8;
		  (n+8)[2] = ((si + bufPos) / 64)>>16;
		  (n+8)[3] = ((si + bufPos) / 64)>>24;

		  // s20_expand16(key, n, keystream);

		  int i, j;
		  uint8_t t[4][4] = {
			{ 'e', 'x', 'p', 'a' },
			{ 'n', 'd', ' ', '1' },
			{ '6', '-', 'b', 'y' },
			{ 't', 'e', ' ', 'k' }
		  };

		  for (i = 0; i < 64; i += 20)
			for (j = 0; j < 4; ++j)
			  keystream[i + j] = t[i / 20][j];

		  for (i = 0; i < 16; ++i) {
			keystream[4+i]  = key[i];
			keystream[44+i] = key[i];
			keystream[24+i] = n[i];
		  }

	  // ____________________
	  // s20_hash(keystream);
	  // --------------------

		//    int i;
		  uint32_t x[16];
		  uint32_t z[16];

		  for (i = 0; i < 16; ++i) {

			// s20_littleendian
			uint8_t* result = keystream + (4 * i);
			x[i] = z[i] = (int16_t)(result[0]+(result[1]<<8)); //  s20_littleendian(seq + (4 * i));
		  }

		  for (i = 0; i < 10; ++i) {
				//    s20_doubleround(z);

				  // ColumnRound
				  // s20_quarterround(&x[0], &x[4], &x[8], &x[12]);

				  z[4] =  z[4]  ^ ROTL(z[0]  + z[12], 7);
				  z[8] =  z[8]  ^ ROTL(z[4]  + z[0], 9);
				  z[12] = z[12] ^ ROTL(z[8]  + z[4], 13);
				  z[0] =  z[0]  ^ ROTL(z[12] + z[8], 18);

				  // s20_quarterround(&x[5], &x[9], &x[13], &x[1]);
				  z[9] =  z[9]  ^ ROTL(z[5]  + z[1], 7);
				  z[13] = z[13] ^ ROTL(z[9]  + z[5], 9);
				  z[1] =  z[1]  ^ ROTL(z[13] + z[9], 13);
				  z[5] =  z[5]  ^ ROTL(z[1]  + z[13], 18);

				  // s20_quarterround(&x[10], &x[14], &x[2], &x[6]);
				  z[14]=  z[14] ^ ROTL(z[10] + z[6], 7);
				  z[2] =  z[2]  ^ ROTL(z[14] + z[10], 9);
				  z[6] =  z[6]  ^ ROTL(z[2]  + z[14], 13);
				  z[10] = z[10] ^ ROTL(z[6]  + z[2], 18);

				  // s20_quarterround(&x[15], &x[3], &x[7], &x[11]);
				  z[3] =  z[3]  ^ ROTL(z[15] + z[11], 7);
				  z[7] =  z[7]  ^ ROTL(z[3]  + z[15], 9);
				  z[11] = z[11] ^ ROTL(z[7]  + z[3], 13);
				  z[15] = z[15] ^ ROTL(z[11] + z[7], 18);

				  // Rowround
				  // s20_quarterround(&y[0], &y[1], &y[2], &y[3]);
				  z[1] = z[1] ^ ROTL(z[0]+  z[3], 7);
				  z[2] = z[2] ^ ROTL(z[1] + z[0], 9);
				  z[3] = z[3] ^ ROTL(z[2] + z[1], 13);
				  z[0] = z[0] ^ ROTL(z[3] + z[2], 18);

				  // s20_quarterround(&y[5], &y[6], &y[7], &y[4]);
				  z[6] = z[6] ^ ROTL(z[5] + z[4], 7);
				  z[7] = z[7] ^ ROTL(z[6] + z[5], 9);
				  z[4] = z[4] ^ ROTL(z[7] + z[6], 13);
				  z[5] = z[5] ^ ROTL(z[4] + z[7], 18);

				  // s20_quarterround(&y[10], &y[11], &y[8], &y[9]);
				  z[11] = z[11] ^ ROTL(z[10] + z[9], 7);
				  z[8] =  z[8]  ^ ROTL(z[11] + z[10], 9);
				  z[9] =  z[9]  ^ ROTL(z[8] +  z[11], 13);
				  z[10] = z[10] ^ ROTL(z[9] +  z[8], 18);

				  // s20_quarterround(&y[15], &y[12], &y[12], &y[14]);
				  z[12] = z[12] ^ ROTL(z[15] + z[14], 7);
				  z[12] = z[12] ^ ROTL(z[12] + z[15], 9);
				  z[14] = z[14] ^ ROTL(z[12] + z[12], 13);
				  z[15] = z[15] ^ ROTL(z[14] + z[12], 18);
			  }

			  for (i = 0; i < 16; ++i) {
				z[i] += x[i];
				// s20_rev_littleendian(seq + (4 * i), z[i]);
				  (keystream + (4 * i))[0] = z[i];
				  (keystream + (4 * i))[1] = z[i] >> 8;
				  (keystream + (4 * i))[2] = z[i] >> 16;
				  (keystream + (4 * i))[3] = z[i] >> 24;
			  }

			}
			buf[bufPos] ^= keystream[(si + bufPos) % 64];
		  }

	  	  (isValid)[threadNr] = true; // Assume we found the key

		  // Validate Crypto Result
		  for (size_t bufPos = 0; bufPos < VERIBUF_SIZE; bufPos++) {
			 if (buf[bufPos] != VERIFICATION_CHAR) {
				(isValid)[threadNr] = false; // We didn't

				// Calculate next key to try...
				int posToKey[] = {13,12,9,8,5,4,1,0};

				for (int i=0; i<8; i++) {
					int idx = keyToIndexMap[key[posToKey[i]]];
					idx++;
					idx %=sizeof(keyChars);
					key[posToKey[i]] = keyChars[idx];

					if (idx!=0) break;
				}
				break;
			}
		  }

		  nrKeysToCalculatePerThread--;
  	  }

}


void stub(char *x){
	x[0]=5;
}

void initializeAndCalculate(uint8_t nonce_hc[8],  char *verificationBuffer_hc) {

    int n=2048*2048;

    char p_key[(KEY_SIZE)*NR_KEYS];
    char *key = p_key;             
    
    for (unsigned long i=0; i<NR_KEYS;i++){
    	calculate16ByteKeyFromIndex(0, key+i*NR_KEYS);
    }

    hipError_t err = hipSuccess;
    
    uint8_t *verifbuf_test_dc = NULL;
    char veribuf_test_local[VERIBUF_SIZE];
         
    err = hipMalloc((void **)&verifbuf_test_dc, VERIBUF_SIZE);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    err = hipMemcpy(verifbuf_test_dc, verificationBuffer_hc, VERIBUF_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy verification buffer from host to gpu (error code %s)!\n",      hipGetErrorString(err));
        exit(EXIT_FAILURE);

    }
    
    
                    
    uint32_t si_hc;
    uint8_t *buf_hc;
    uint32_t buflen_hc;
    
    uint8_t *key_dc;                       
    uint8_t *nonce_dc;
    uint32_t si_dc = 0;
    uint8_t *buf_dc;
    uint32_t buflen_dc;
    
    
    err = hipMalloc((void **)&key_dc, (KEY_SIZE)*NR_KEYS);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for key (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    err = hipMalloc((void **)&nonce_dc, 8);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for nonce (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    

    
    
    err = hipMemcpy(nonce_dc, nonce_hc, 8, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy nonce from host to gpu (error code %s)!\n",      hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    bool result_hc[NR_KEYS];
    bool *result_dc;
    
    err = hipMalloc((void **)&result_dc, sizeof(bool)*NR_KEYS);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for isValid Result (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    bool keyFound = false;
    
    unsigned long nrOfKeysPerThreadAndCall = 1; // pow(2*26+10,8)/NR_THREADS
    
    do {
        
        err = hipMemcpy(key_dc, (uint8_t *) key, (KEY_SIZE)*NR_KEYS, hipMemcpyHostToDevice);        
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy key from host to gpu (error code %s)!\n",      hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
                                
        gpu_crypt_and_validate<<<NR_BLOCKS, NR_THREADS>>>(key_dc, 
                                         nonce_dc, 
                                         si_dc, 
                                         verifbuf_test_dc, 
                                         VERIBUF_SIZE, 
                                         result_dc,n, 
										 nrOfKeysPerThreadAndCall										 
        								  );
        
        
        
        err = hipGetLastError();    
        if (err != hipSuccess)
        {
            fprintf(stderr, "error starting thread on gpu (error code %s)!\n",      hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    
        
        err = hipMemcpy(&result_hc, 
                         result_dc, 
                         sizeof(bool)*NR_KEYS, 
                         hipMemcpyDeviceToHost);   
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy memory from device to host (error code %s)!\n",      hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        
        err = hipMemcpy((uint8_t *) key, key_dc, (KEY_SIZE)*NR_KEYS, hipMemcpyDeviceToHost);        
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy key from host to gpu (error code %s)!\n",      hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
        
        for (int i=0; i<NR_KEYS;i++) {
            if (result_hc[i]) {
                printf("Key found:\r\n");
                for (int j=0; j<KEY_SIZE; j++) {
                    printf("%c", key[(KEY_SIZE)*i+j]);
                }
                printf("\r\n");
                keyFound = true;
            }
        }

        printf("Next round\r\n");

        // Calculate next keys for next round...
        for (unsigned long i=0;i<(KEY_SIZE)*NR_KEYS;i++) {
        	char *currentKey = (key+i*KEY_SIZE); 
        	nextKey16Byte(currentKey);
        }
        
        
        
    } while (!keyFound);
    
    
    
    

    // Free device global memory
     err = hipFree(result_dc);
    
    
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device memory (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    
    
    err = hipFree(verifbuf_test_dc);
    
    
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device memory (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
}






